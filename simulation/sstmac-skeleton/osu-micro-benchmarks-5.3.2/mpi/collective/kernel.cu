
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2002-2016 the Network-Based Computing Laboratory
 * (NBCL), The Ohio State University.
 *
 * Contact: Dr. D. K. Panda (panda@cse.ohio-state.edu)
 *
 * For detailed copyright and licensing information, please refer to the
 * copyright file COPYRIGHT in the top level OMB directory.
 */

__global__ 
void compute_kernel(float a, float * x, float * y, int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    int count = 0;
    
    if (i < N) {
        for(count=0; count < (N/8); count++) { 
            y[i] = a * x[i] + y[i];
        }
    }
}   

extern "C" 
void 
call_kernel(float a, float * d_x, float * d_y, int N, hipStream_t * stream)
{
    compute_kernel<<<(N+255)/256, 256, 0, *stream>>>(a, d_x, d_y, N);
}


